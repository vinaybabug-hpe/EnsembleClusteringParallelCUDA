#include "hip/hip_runtime.h"
#include <iostream>
#include "rsymsol.h"
#include "arrssym.h"
#include <fstream>
#include <cstdlib>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include "hipsparse.h"
#include "hip/hip_runtime.h"
#include <vector>
#include "timer.h"
#include "kmeans.h"
#include "common/wrapper.h"
#include "common/wrapperFuncs.h"

using namespace std;

int CUDA_MULT(float *x, float *y, hipsparseHandle_t& handle, hipsparseStatus_t& status, hipsparseMatDescr_t& descr, int n, int nnz, thrust::device_vector<int>& csrRowPtr, thrust::device_vector<int>& cooColIndex, thrust::device_vector<float>& cooVal, thrust::device_vector<float>& tmpx, thrust::device_vector<float>& tmpy){
	float fone = 1.0;
	float fzero = 0.0;
	hipMemcpy(thrust::raw_pointer_cast(tmpx.data()), x, n*sizeof(float), hipMemcpyHostToDevice);
	status = hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
			n, n, nnz, &fone, 
			descr, 
			thrust::raw_pointer_cast(cooVal.data()), 
			thrust::raw_pointer_cast(csrRowPtr.data()) , thrust::raw_pointer_cast(cooColIndex.data()),
			thrust::raw_pointer_cast(tmpx.data()), &fzero, 
			thrust::raw_pointer_cast(tmpy.data()));
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("hipsparseScsrmv Failed");
		return 1;
	}
	hipMemcpy(y, thrust::raw_pointer_cast(tmpy.data()), n*sizeof(float), hipMemcpyDeviceToHost);
	return 0;

}


void random_labels(thrust::device_vector<int>& labels, int n, int k) {
	thrust::host_vector<int> host_labels(n);
	for(int i = 0; i < n; i++) {
		host_labels[i] = rand() % k;
	}
	labels = host_labels;
}

void regular_labels(thrust::device_vector<int>& labels, int n, int k) {
	// Initialize by assigning nodes that are close in indexing order with the same label.
	thrust::host_vector<int> host_labels(n);
	int l = n/k;
	int count = 0;
	int cur = 0;
	for(int i = 0; i < n; i++) {
		host_labels[i] = cur;
		count++;
		if(count > l) {
			cur++;
			count = 0;
		}
	}
	labels = host_labels;
}

void fastsc(int nrows, int ncols, int numclusters,float *distmatrix, int* idxs)
{

	int n = nrows;
	int k = numclusters;
	int nnz = (nrows * nrows) - nrows;
	// CUDA related variables
	hipError_t err;

	thrust::host_vector<int> row(nnz), col(nnz);

	// Initialize the degree
	thrust::host_vector<float> degree(n, 0.0);

	// For unweighted graphs, edge weights are initilized to 1.0. Otherwise, revise the code to the specific graph representation.
	thrust::host_vector<float> val(nnz, 1.0);

	int count = 0;

	//	for (int i = 0; i < nelements; i++){
	//		for (int j = 0; j < i; j++){
	//			distmatrix_h [i * nrows + j] = distmatrix_h [j * nrows + i] =  distmatrix_m[TRI_COUNT(i)+j];
	////			printf("%.4f ", distmatrix_m[TRI_COUNT(i)+j]);
	//		}
	////		printf("\n");
	//	}


	for(int crow=0; crow<nrows; crow++){
		for(int ccol=0; ccol<crow; ccol++){

				row[count] = crow;
				col[count] = ccol;
				val[count] =  distmatrix[TRI_COUNT(crow)+ccol];
				degree[row[count]] = degree[row[count]] + val[count];
				//					cout<<"[" <<count<<"]"<< row[count]<< " "<< col[count]<< " "<< val[count]<< " "<< degree[row[count]]<< " "<<endl;
				count++;

				row[count] = ccol;
				col[count] = crow;
				val[count] =  distmatrix[TRI_COUNT(crow)+ccol];
				degree[row[count]] = degree[row[count]] + val[count];
				count++;

		}
	}


//		cout<<"Start computing normalized Graph Laplacian..."<<endl;
//		for(int i = 0; i < n; ++i) {
//			if (degree[i] < 1e-8) {
//				cout<<"Node " <<i<<" is an isolated node"<<endl;
//				cout<<"Please eliminate isolated nodes and try again!"<<endl;
//				exit(1);
//			}
//		}

		thrust::host_vector<float> degree_sqrt(n);

		// Normlize the edge weight of <i, j> by 1.0/sqrt(degree[i] * degree[j])
		for(int i = 0; i < n; ++i) {
			degree_sqrt[i] = sqrt(degree[i]);
		}

		for(int i = 0; i < nnz; ++i) {
			val[i] = val[i] / (degree_sqrt[col[i]] * degree_sqrt[row[i]]);
		}

//		cout<<"Computing normalized Graph Laplacian completed"<<endl;
//		cout<<"Start computing the first smallest "<< k <<" eigenvectors..."<<endl;
		thrust::device_vector<int> cooRowIndex = row;
		thrust::device_vector<int> cooColIndex = col;
		thrust::device_vector<float> cooVal = val;
		hipsparseStatus_t status;
		hipsparseHandle_t handle=0;
		hipsparseMatDescr_t descr=0;
		status= hipsparseCreate(&handle);
		status= hipsparseCreateMatDescr(&descr);
		if (status != HIPSPARSE_STATUS_SUCCESS) {
			printf("Matrix descriptor initialization failed");
			return;
		}
		hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
		thrust::device_vector<int> csrRowPtr(n+1);

		status= hipsparseXcoo2csr(handle,thrust::raw_pointer_cast(cooRowIndex.data()),nnz,n,
				thrust::raw_pointer_cast(csrRowPtr.data()),HIPSPARSE_INDEX_BASE_ZERO);
		if (status != HIPSPARSE_STATUS_SUCCESS) {
			printf("Conversion from COO to CSR format failed");
			return;
		}
		thrust::device_vector<float> tmpx(n);
		thrust::device_vector<float> tmpy(n);
		ARrcSymStdEig<float> prob(n, k, "LM");
		while (!prob.ArnoldiBasisFound()) {
			prob.TakeStep();
			if ((prob.GetIdo() == 1)||(prob.GetIdo() == -1)) {
				CUDA_MULT(prob.GetVector(), prob.PutVector(), handle, status, descr, n, nnz, csrRowPtr, cooColIndex, cooVal, tmpx, tmpy);
			}
		}

		// Finding eigenvalues and eigenvectors.
		prob.FindEigenvectors();
		// Printing eigenvalue solution.
		// Solution(prob);

		hipsparseDestroy(handle);

//		cout<<"Completed computing the first smallest k eigenvectors!"<<endl;

		// Extract eigenvectors.
		// Rearrange the order such that values between i * k and (i+1)*k-1 are eigenmap for node indexed by i
//		cout<<"Start kmeans clustering algorithm on the k eigenvectors..."<<endl;
		// TODO: Re-write kmeans as this implementation is very slow! :0

		/*
		 * Variables to keep track of memory used on device.
		 * Depending on the current memory usage we need to restrict
		 * number of threads.
		 */
		ncols = k;



		float* weight;
		float *eigen_objects, *eigen_objects_host;

		size_t memDataSz = 0, memWtSz = 0, memIdxSz=0;

		memDataSz = nrows * ncols * sizeof(float);
		CUDA_CHECK_RETURN(hipMalloc(&eigen_objects, memDataSz));
		assert(eigen_objects != NULL);
		eigen_objects_host = (float*) malloc(memDataSz);
		assert(eigen_objects_host != NULL);


		for (int i = 0; i < n; ++i) {
			for (int j = 0; j < k; ++j) {

				eigen_objects_host[i*k + j] = prob.Eigenvector(j, i);

			}
		}

		CUDA_CHECK_RETURN(hipMemcpy(eigen_objects, eigen_objects_host,  memDataSz, hipMemcpyHostToDevice));

		memWtSz = ncols * sizeof(float);

		CUDA_CHECK_RETURN(hipMalloc(&weight, memWtSz));

		assert(weight != NULL);

		// wrap raw pointer with a device_ptr
		thrust::device_ptr<float> weight_ptr(weight);

		// use device_ptr in thrust algorithms
		thrust::fill(weight_ptr, weight_ptr + ncols, (float) 1);


		// bind texture to buffer
		// create texture object
		hipResourceDesc resDescData;
		memset(&resDescData, 0, sizeof(resDescData));
		resDescData.resType = hipResourceTypeLinear;
		resDescData.res.linear.devPtr = eigen_objects;
		resDescData.res.linear.desc.f = hipChannelFormatKindFloat;
		resDescData.res.linear.desc.x = 32; // bits per channel
		resDescData.res.linear.sizeInBytes = memDataSz;

		hipResourceDesc resDescWt;
		memset(&resDescWt, 0, sizeof(resDescWt));
		resDescWt.resType = hipResourceTypeLinear;
		resDescWt.res.linear.devPtr = weight;
		resDescWt.res.linear.desc.f = hipChannelFormatKindFloat;
		resDescWt.res.linear.desc.x = 32; // bits per channel
		resDescWt.res.linear.sizeInBytes = memWtSz;

		hipTextureDesc texDescData;
		memset(&texDescData, 0, sizeof(texDescData));
		texDescData.readMode = hipReadModeElementType;

		hipTextureDesc texDescWt;
		memset(&texDescWt, 0, sizeof(texDescWt));
		texDescWt.readMode = hipReadModeElementType;

		// create texture object: we only have to do this once!
		hipTextureObject_t texData = 0;
		hipCreateTextureObject(&texData, &resDescData, &texDescData, NULL);
		if (hipSuccess != (err = hipGetLastError()))
		{
			printf("\n6. @texData error: %s\n",	hipGetErrorString(err));
			return;
		}

		hipTextureObject_t texWt = 0;
		hipCreateTextureObject(&texWt, &resDescWt, &texDescWt, NULL);
		if (hipSuccess != (err = hipGetLastError()))
		{
			printf("\n8. @texWt error: %s\n", hipGetErrorString(err));
			return;
		}

		int *maxGridSize;
		int *maxThreadsPerBlock;
		hipDeviceProp_t prop;
		int device;
		hipGetDevice(&device);
		hipGetDeviceProperties(&prop, device);

		CUDA_CHECK_RETURN(hipMalloc(&maxGridSize, sizeof(int)));
		CUDA_CHECK_RETURN(hipMalloc(&maxThreadsPerBlock, sizeof(int)));
		CUDA_CHECK_RETURN(hipMemcpy(maxGridSize, &prop.maxGridSize[1], sizeof(int), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(maxThreadsPerBlock, &prop.maxThreadsPerBlock, sizeof(int), hipMemcpyHostToDevice));

		//Apply K-means algorithm on the eigenvectors
		int iterations = 100;
		double threshold=0.000001;
		// The dimension of each point is equal to the number of desired clusters.

		cuda_kmedians('e', eigen_objects_host, ncols, nrows, k, threshold, idxs, &iterations, texData, texWt, maxGridSize, maxThreadsPerBlock);

		// destroy texture object
		hipDestroyTextureObject(texData);
		hipDestroyTextureObject(texWt);
		CUDA_CHECK_RETURN(hipFree(eigen_objects));
		CUDA_CHECK_RETURN(hipFree(weight));
		free(eigen_objects_host);
		CUDA_CHECK_RETURN(hipFree(maxGridSize));
		CUDA_CHECK_RETURN(hipFree(maxThreadsPerBlock));
		return;
}

void fastsc_full_distmat(int nrows, int ncols, int numclusters,float *distmatrix, int* idxs)
{

	int n = nrows;
	int k = numclusters;
	int nnz = (nrows * nrows) - nrows;
	// CUDA related variables
	hipError_t err;

	thrust::host_vector<int> row(nnz), col(nnz);

	// Initialize the degree
	thrust::host_vector<float> degree(n, 0.0);

	// For unweighted graphs, edge weights are initilized to 1.0. Otherwise, revise the code to the specific graph representation.
	thrust::host_vector<float> val(nnz, 1.0);

	int count = 0;

	//	for (int i = 0; i < nelements; i++){
	//		for (int j = 0; j < i; j++){
	//			distmatrix_h [i * nrows + j] = distmatrix_h [j * nrows + i] =  distmatrix_m[TRI_COUNT(i)+j];
	////			printf("%.4f ", distmatrix_m[TRI_COUNT(i)+j]);
	//		}
	////		printf("\n");
	//	}


	for(int crow=0; crow<nrows; crow++){
		for(int ccol=0; ccol<ncols; ccol++){

			if(crow != ccol){
				row[count] = crow;
				col[count] = ccol;
				val[count] =  distmatrix[crow * ncols + ccol];
				degree[row[count]] = degree[row[count]] + val[count];
				//					cout<<"[" <<count<<"]"<< row[count]<< " "<< col[count]<< " "<< val[count]<< " "<< degree[row[count]]<< " "<<endl;
				count++;
			}

		}
	}


//		cout<<"Start computing normalized Graph Laplacian..."<<endl;
//		for(int i = 0; i < n; ++i) {
//			if (degree[i] < 1e-8) {
//				cout<<"Node " <<i<<" is an isolated node"<<endl;
//				cout<<"Please eliminate isolated nodes and try again!"<<endl;
//				exit(1);
//			}
//		}
		thrust::host_vector<float> degree_sqrt(n);

		// Normlize the edge weight of <i, j> by 1.0/sqrt(degree[i] * degree[j])
		for(int i = 0; i < n; ++i) {
			degree_sqrt[i] = sqrt(degree[i]);
		}

		for(int i = 0; i < nnz; ++i) {
			val[i] = val[i] / (degree_sqrt[col[i]] * degree_sqrt[row[i]]);
		}

//		cout<<"Computing normalized Graph Laplacian completed"<<endl;
//		cout<<"Start computing the first smallest "<< k <<" eigenvectors..."<<endl;
		thrust::device_vector<int> cooRowIndex = row;
		thrust::device_vector<int> cooColIndex = col;
		thrust::device_vector<float> cooVal = val;
		hipsparseStatus_t status;
		hipsparseHandle_t handle=0;
		hipsparseMatDescr_t descr=0;
		status= hipsparseCreate(&handle);
		status= hipsparseCreateMatDescr(&descr);
		if (status != HIPSPARSE_STATUS_SUCCESS) {
			printf("Matrix descriptor initialization failed");
			return;
		}
		hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
		thrust::device_vector<int> csrRowPtr(n+1);

		status= hipsparseXcoo2csr(handle,thrust::raw_pointer_cast(cooRowIndex.data()),nnz,n,
				thrust::raw_pointer_cast(csrRowPtr.data()),HIPSPARSE_INDEX_BASE_ZERO);
		if (status != HIPSPARSE_STATUS_SUCCESS) {
			printf("Conversion from COO to CSR format failed");
			return;
		}
		thrust::device_vector<float> tmpx(n);
		thrust::device_vector<float> tmpy(n);
		ARrcSymStdEig<float> prob(n, k, "LM");
		while (!prob.ArnoldiBasisFound()) {
			prob.TakeStep();
			if ((prob.GetIdo() == 1)||(prob.GetIdo() == -1)) {
				CUDA_MULT(prob.GetVector(), prob.PutVector(), handle, status, descr, n, nnz, csrRowPtr, cooColIndex, cooVal, tmpx, tmpy);
			}
		}

		// Finding eigenvalues and eigenvectors.
		prob.FindEigenvectors();
		// Printing eigenvalue solution.
		// Solution(prob);

		hipsparseDestroy(handle);

//		cout<<"Completed computing the first smallest k eigenvectors!"<<endl;

		// Extract eigenvectors.
		// Rearrange the order such that values between i * k and (i+1)*k-1 are eigenmap for node indexed by i
//		cout<<"Start kmeans clustering algorithm on the k eigenvectors..."<<endl;
		// TODO: Re-write kmeans as this implementation is very slow! :0

		/*
		 * Variables to keep track of memory used on device.
		 * Depending on the current memory usage we need to restrict
		 * number of threads.
		 */
		ncols = k;


		float* weight;
		float *eigen_objects, *eigen_objects_host;

		size_t memDataSz = 0, memWtSz = 0, memIdxSz=0;

		memDataSz = nrows * ncols * sizeof(float);
		CUDA_CHECK_RETURN(hipMalloc(&eigen_objects, memDataSz));
		assert(eigen_objects != NULL);
		eigen_objects_host = (float*) malloc(memDataSz);
		assert(eigen_objects_host != NULL);


		for (int i = 0; i < n; ++i) {
			for (int j = 0; j < k; ++j) {

				eigen_objects_host[i*k + j] = prob.Eigenvector(j, i);

			}
		}

		CUDA_CHECK_RETURN(hipMemcpy(eigen_objects, eigen_objects_host,  memDataSz, hipMemcpyHostToDevice));

		memWtSz = ncols * sizeof(float);

		CUDA_CHECK_RETURN(hipMalloc(&weight, memWtSz));

		assert(weight != NULL);

		// wrap raw pointer with a device_ptr
		thrust::device_ptr<float> weight_ptr(weight);

		// use device_ptr in thrust algorithms
		thrust::fill(weight_ptr, weight_ptr + ncols, (float) 1);


		// bind texture to buffer
		// create texture object
		hipResourceDesc resDescData;
		memset(&resDescData, 0, sizeof(resDescData));
		resDescData.resType = hipResourceTypeLinear;
		resDescData.res.linear.devPtr = eigen_objects;
		resDescData.res.linear.desc.f = hipChannelFormatKindFloat;
		resDescData.res.linear.desc.x = 32; // bits per channel
		resDescData.res.linear.sizeInBytes = memDataSz;

		hipResourceDesc resDescWt;
		memset(&resDescWt, 0, sizeof(resDescWt));
		resDescWt.resType = hipResourceTypeLinear;
		resDescWt.res.linear.devPtr = weight;
		resDescWt.res.linear.desc.f = hipChannelFormatKindFloat;
		resDescWt.res.linear.desc.x = 32; // bits per channel
		resDescWt.res.linear.sizeInBytes = memWtSz;

		hipTextureDesc texDescData;
		memset(&texDescData, 0, sizeof(texDescData));
		texDescData.readMode = hipReadModeElementType;

		hipTextureDesc texDescWt;
		memset(&texDescWt, 0, sizeof(texDescWt));
		texDescWt.readMode = hipReadModeElementType;

		// create texture object: we only have to do this once!
		hipTextureObject_t texData = 0;
		hipCreateTextureObject(&texData, &resDescData, &texDescData, NULL);
		if (hipSuccess != (err = hipGetLastError()))
		{
			printf("\n6. @texData error: %s\n",	hipGetErrorString(err));
			return;
		}

		hipTextureObject_t texWt = 0;
		hipCreateTextureObject(&texWt, &resDescWt, &texDescWt, NULL);
		if (hipSuccess != (err = hipGetLastError()))
		{
			printf("\n8. @texWt error: %s\n", hipGetErrorString(err));
			return;
		}

		int *maxGridSize;
		int *maxThreadsPerBlock;
		hipDeviceProp_t prop;
		int device;
		hipGetDevice(&device);
		hipGetDeviceProperties(&prop, device);

		CUDA_CHECK_RETURN(hipMalloc(&maxGridSize, sizeof(int)));
		CUDA_CHECK_RETURN(hipMalloc(&maxThreadsPerBlock, sizeof(int)));
		CUDA_CHECK_RETURN(hipMemcpy(maxGridSize, &prop.maxGridSize[1], sizeof(int), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(maxThreadsPerBlock, &prop.maxThreadsPerBlock, sizeof(int), hipMemcpyHostToDevice));

		//Apply K-means algorithm on the eigenvectors
		int iterations = 100;
		double threshold=0.000001;
		// The dimension of each point is equal to the number of desired clusters.

		cuda_kmedians('e', eigen_objects_host, ncols, nrows, k, threshold, idxs, &iterations, texData, texWt, maxGridSize, maxThreadsPerBlock);

		// destroy texture object
		hipDestroyTextureObject(texData);
		hipDestroyTextureObject(texWt);
		CUDA_CHECK_RETURN(hipFree(eigen_objects));
		CUDA_CHECK_RETURN(hipFree(weight));
		free(eigen_objects_host);
		CUDA_CHECK_RETURN(hipFree(maxGridSize));
		CUDA_CHECK_RETURN(hipFree(maxThreadsPerBlock));

		return;
}

void hello_fastsc(){
	printf("\n~~~HELLO FROM FAST SC~~~\n");
}

int __main(int argc, char* argv[]) {
	if(argc < 5) {
		cout<<"Not enough input arguments!"<<endl;
		cout<<"The input format is: " <<endl;
		cout<<"1. Filename"<<endl;
		cout<<"2. Number of nodes n"<<endl;
		cout<<"3. Number of clusters k"<<endl;
		cout<<"4. Output labeling file"<<endl;
		exit(1);
	}
	// The graph is represented in edgelist format.
	// Each row represent the edge between <i, j>. 
	// For undirected graphs, both <i, j> and <j, i> need to be included in the file.
	// Nodes are indexed from 0 to n-1 with no isolated nodes.
	ifstream infile(argv[1]);
	if(!infile) {
		cout<<"wrong input file"<<endl;
		return 1;
	}   
	int n = atoi(argv[2]);
	int k = atoi(argv[3]);
	string line;
	int nnz = 0;

	// Get the number of edges
	while(getline(infile, line)) nnz++;
	thrust::host_vector<int> row(nnz), col(nnz);

	// Initialize the degree
	thrust::host_vector<float> degree(n, 0.0);

	// For unweighted graphs, edge weights are initilized to 1.0. Otherwise, revise the code to the specific graph representation.
	thrust::host_vector<float> val(nnz, 1.0);
	infile.close();
	infile.open(argv[1]);
	cout<<"Start loading data..."<<endl;
	for(int i = 0; i < nnz; ++i) {
		infile>>row[i]>>col[i];
		if (row[i] >= n || col[i] >= n) {
			cout<<"Index exceed the dimension. Please check the right number of nodes"<<endl;
			exit(1);
		}
		// If the input graph is weighted, change it to
		//infile>>row[i]>>col[i]>>val[i];
		degree[row[i]] = degree[row[i]] + val[i];
	}
	infile.close();
	cout<<"Loading data completed!"<<endl;

	cout<<"Start computing normalized Graph Laplacian..."<<endl;
	for(int i = 0; i < n; ++i) {
		if (degree[i] < 1e-8) {
			cout<<"Node " <<i<<" is an isolated node"<<endl;
			cout<<"Please eliminate isolated nodes and try again!"<<endl;
			exit(1);
		}
	}
	thrust::host_vector<float> degree_sqrt(n);

	// Normlize the edge weight of <i, j> by 1.0/sqrt(degree[i] * degree[j])
	for(int i = 0; i < n; ++i) {
		degree_sqrt[i] = sqrt(degree[i]);
	}

	for(int i = 0; i < nnz; ++i) {
		val[i] = val[i] / (degree_sqrt[col[i]] * degree_sqrt[row[i]]);
	}

	cout<<"Computing normalized Graph Laplacian completed"<<endl;
	cout<<"Start computing the first smallest k eigenvectors..."<<endl;
	thrust::device_vector<int> cooRowIndex = row;
	thrust::device_vector<int> cooColIndex = col;
	thrust::device_vector<float> cooVal = val;
	hipsparseStatus_t status;
	hipsparseHandle_t handle=0;
	hipsparseMatDescr_t descr=0;
	status= hipsparseCreate(&handle);
	status= hipsparseCreateMatDescr(&descr);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("Matrix descriptor initialization failed");
		return 1;
	}
	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
	thrust::device_vector<int> csrRowPtr(n+1);

	status= hipsparseXcoo2csr(handle,thrust::raw_pointer_cast(cooRowIndex.data()),nnz,n,
			thrust::raw_pointer_cast(csrRowPtr.data()),HIPSPARSE_INDEX_BASE_ZERO);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("Conversion from COO to CSR format failed");
		return 1;
	}
	thrust::device_vector<float> tmpx(n);
	thrust::device_vector<float> tmpy(n);
	ARrcSymStdEig<float> prob(n, k, "LM");
	while (!prob.ArnoldiBasisFound()) {
		prob.TakeStep();
		if ((prob.GetIdo() == 1)||(prob.GetIdo() == -1)) {
			CUDA_MULT(prob.GetVector(), prob.PutVector(), handle, status, descr, n, nnz, csrRowPtr, cooColIndex, cooVal, tmpx, tmpy);
		}
	}

	// Finding eigenvalues and eigenvectors.
	prob.FindEigenvectors();
	// Printing eigenvalue solution.
	// Solution(prob);

	cout<<"Completed computing the first smallest k eigenvectors!"<<endl;

	// Extract eigenvectors. 
	// Rearrange the order such that values between i * k and (i+1)*k-1 are eigenmap for node indexed by i
	cout<<"Start kmeans clustering algorithm on the k eigenvectors..."<<endl;
	thrust::host_vector<float> eigenvectors_h(n*k);
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < k; ++j) {
			eigenvectors_h[i*k + j] = prob.Eigenvector(j, i);
		}
	}

	//Apply K-means algorithm on the eigenvectors
	int iterations = 100;
	// The dimension of each point is equal to the number of desired clusters.
	int d = k; 
	thrust::device_vector<float> eigenvectors_d = eigenvectors_h; 
	thrust::device_vector<int> labels(n);
	thrust::device_vector<float> centroids(k * d); 
	thrust::device_vector<float> distances(n);
	// Randomly initialize the labels. (You can also try the regular_labels)
	random_labels(labels, n, k);
	kmeans::kmeans(iterations, n, d, k, eigenvectors_d, labels, centroids, distances);
	cout<<"Completed kmeans clustering algorithm on the k eigenvectors!"<<endl;
	cout<<"Start output clustering results..."<<endl;
	ofstream outfile(argv[4]);
	outfile<<"Node ID" <<' ' <<"Label"<<endl;
	for(int i = 0; i < n; ++i){ 
		outfile<<i<<' '<<labels[i]<<endl;
	} 
	outfile.close();
	cout<<"Completed output clustering results!"<<endl;
	return 0;
}
