#include "hip/hip_runtime.h"
/*
 =====================================================================================================================
 Name        : clster_ensemble_create.cu
 Author      : Vinay B Gavirangaswamy
 Version     : 1.0
 Copyright   :  This file is part of application to do "Ensemble Clustering Analysis on CUDA".

    			"Ensemble Clustering Analysis on CUDA" is free software: you can redistribute it and/or modify
    			it under the terms of the GNU General Public License as published by the Free Software Foundation,
    			either version 3 of the License, or (at your option) any later version.

    			"Ensemble Clustering Analysis on CUDA" is distributed in the hope that it will be useful, but
    			WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
    			PARTICULAR PURPOSE.  See the GNU General Public License for more details.

    			You should have received a copy of the GNU General Public License along with
    			"Ensemble Cluster Analysis on CUDA".  If not, see <http://www.gnu.org/licenses/>.

 Description :
 =====================================================================================================================
 */

#include <iostream>
#include <fstream>
#include <cstdio>
#include <cstdlib>
#include <regex.h>
#include <string>
#include <cstring>
#include <cassert>
#include <cfloat>
#include <cmath>
#include <numeric>
#include <stdlib.h>
#include <iostream>


#include "mat.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>

#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>

#include "common/wrapper.h"
#include "common/wrapperFuncs.h"
#include "common/distCalcMthds.h"
#include "common/clustLib.h"
#include "common/cluster_data_outliers.h"
#include "common/cluster_util_bootpartition2partition.h"
#include "common/cluster_ensemble2cam.h"
#include "common/cluster_util_indices2centers.h"
#include "common/indices_count.h"
#include "common/cluster_util_ssw.h"




using namespace std;
////////////////////////////////////////////////////////////////////////////////
// Main entry point.
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    // NON MEX related variables
	char* en_name = "Ensemble Clustering Results using CUDA";
	char* en_desc;
	char *output_file;
	int ifield, nfields, field_num;
    int nStructElems = 0;
    int VERBOSE_Q, INCLUDE_REPsQ, INCLUDE_CENTERsQ;
    int nBootstrap = 0;
    int nReps = 0;
    char *name;
    double *kList;
    char **clustList;
    char **modelList;
    int nKList, nModelList;
    int m_x, n_x;
    int *bootIdxs;
    double *x_actual_data;
    float *x_transpose;
    int count_x_transpose;
	char *distmetric;
	char *centerfun;
	char *linkcode;

	char *outlierMetric;
	float outlierCutoff;

	int num_total_partitions = 0;
	int *data_partition_indices;

    // MEX related variables
    MATFile *mfPtr, *mfPtr2; /* MAT-file pointer */
    mxArray *aPST, *aX, *aBootIdxs;  /* mxArray pointer */
    mxArray *tmp, *pst_ensemble, *pst_extract;
	const mxArray *cell_element_ptr;
	mxArray *output_cell_array_ptr;

	// CUDA related variables
	hipError_t err;

    if (argc < 4){
		cout << "Not enough input arguments!" << endl;
		cout << "The input format is: " << endl;
		cout << "1. PST .mat Filename" << endl;
		cout << "2. DATA .mat Filename" << endl;
		cout << "3. Ensemble Solution Description" << endl;
		cout << "4. Output Filename (.mat)" << endl;
		exit(1);
	}

	// Find/set the device.
	int device_count = 0, device = -1;

	checkCudaErrors(hipGetDeviceCount(&device_count));
	hipDeviceProp_t properties;

	for (int i = 0; i < device_count; ++i) {
		checkCudaErrors(hipGetDeviceProperties(&properties, i));
		if (properties.major > 3
				|| (properties.major == 3 && properties.minor >= 5)) {
			device = i;
			printf("\n2. Running on GPU  %d (%s)", i, properties.name);


			//			std::cout << "Running on GPU " << i << " (" << properties.name<< ")" << std::endl;
			break;
		}
		printf("\n3. GPU  %d (%s) does not support CUDA Dynamic Parallelism", i, properties.name);

		//		std::cout << "GPU " << i << " (" << properties.name<< ") does not support CUDA Dynamic Parallelism" << std::endl;
	}
	if (device == -1) {
		printf("\n4. CUDA Ensemble requires GPU devices with compute SM 3.5 or higher.  Exiting...");
		//		std::cerr<< "cdpSimplePrint requires GPU devices with compute SM 3.5 or higher.  Exiting..."<< std::endl;
		exit (EXIT_WAIVED);
	}
	hipSetDevice(device);

    // TODO: Debug code remove later!
//    for(int count = 0; count < argc; count++){
//    	cout << "argv["<<count<<"]: " << argv[count] << endl;
//    }

    mfPtr = matOpen(argv[1], "r");
    if (mfPtr == NULL) {
        printf("Error opening file %s\n", argv[1]);
        return(EXIT_FAILURE);
    }

    aPST = matGetVariable(mfPtr, MEX_STRUCT_PST_NME);
    if (aPST == NULL) {
        printf("mxArray not found: %s\n", MEX_STRUCT_PST_NME);
        return(EXIT_FAILURE);
    }

    mfPtr2 = matOpen(argv[2], "r");
    if (mfPtr2 == NULL) {
    	printf("Error opening file %s\n", argv[2]);
    	return(EXIT_FAILURE);
    }

    aX = matGetVariable(mfPtr2, MEX_STRUCT_DATA_NME);
    if (aX == NULL) {
    	printf("mxArray not found: %s\n", MEX_STRUCT_PST_NME);
    	return(EXIT_FAILURE);
    }

    en_desc = argv[3];

    output_file = argv[4];

	nfields = mxGetNumberOfFields(aPST);
	nStructElems = mxGetNumberOfElements(aPST);
	/* check proper input and output */
	if (nStructElems != 1){
		printf("ENCLUST CUDA: Multiple PST structures \n Only one input is required.");
	}

	tmp = mxGetField(aPST, 0, MEX_STRUCT_PST_FIELD_VERBOSE);
	VERBOSE_Q = mxGetScalar(tmp);
	field_num = mxGetFieldNumber(aPST, MEX_STRUCT_PST_FIELD_NAME);
	tmp = mxGetFieldByNumber(aPST, 0, field_num);
	name = mxArrayToString(tmp);

	outlierCutoff = mxGetScalar(
				mxGetField(aPST, 0, MEX_STRUCT_OUTLIERCUTOFF_NME));

	outlierMetric = mxArrayToString(mxGetField(aPST, 0, MEX_STRUCT_OUTLIERMETRIC_NME));

//	cout<<"\nNO PROBLEM TILL HERE"<<endl;

	pst_ensemble = mxGetField(aPST, 0, MEX_STRUCT_PST_FIELD_ENSEMBLE);

	/**
	 * Extract data from ENSEMBLE struct
	 * ENSEMBLE is inside PST
	 */
	INCLUDE_REPsQ = mxGetScalar(
			mxGetField(pst_ensemble, 0, MEX_STRUCT_PST_FIELD_ENSEMBLE_INCLUDEREPSQ));
	INCLUDE_CENTERsQ = mxGetScalar(
			mxGetField(pst_ensemble, 0, MEX_STRUCT_PST_FIELD_ENSEMBLE_INCLUDECENTERSQ));
	nBootstrap = mxGetScalar(mxGetField(pst_ensemble, 0, MEX_STRUCT_PST_FIELD_ENSEMBLE_NBOOTSTRAPS));

	if (INCLUDE_REPsQ == 1) {
		nReps = mxGetScalar(mxGetField(pst_ensemble, 0, MEX_STRUCT_PST_FIELD_ENSEMBLE_NREPS));
	} else {
		nReps = 1;
	}

	tmp = mxGetField(pst_ensemble, 0, MEX_STRUCT_PST_FIELD_ENSEMBLE_KLIST);
	nKList = mxGetN(tmp);
	kList = mxGetPr(tmp);

	/*
	 * Get list of model and cluster to run.
	 */

	tmp = mxGetField(pst_ensemble, 0, MEX_STRUCT_PST_FIELD_ENSEMBLE_MODELLIST);
	nModelList = mxGetNumberOfElements(tmp);

	modelList = (char**) malloc(nModelList * sizeof(char*));

	for (int j = 0; j < nModelList; j++) {
		cell_element_ptr = mxGetCell(tmp, j);
		modelList[j] = mxArrayToString(cell_element_ptr);
		/*fprintf(ofp, "%s\n", modelList[j]);*/
	}


	m_x = mxGetM(aX);
	n_x = mxGetN(aX);
	x_actual_data = mxGetPr(aX);

    // size of memory required to store the matrix
    size_t mem_size = static_cast<size_t>(sizeof(float) * m_x* n_x);

    /**
     * Make sure we have enough memory on device. The storage of half-diagonal matrix is not necessary
     * but some clustering algorithm and ensemble process is still programmed using that. Time shortage :(
     *
     */
    if ((
    	(sizeof(float) * m_x* n_x) +
    	(sizeof(int) * m_x* n_x) +
    	(sizeof(float) * n_x) +
    	(sizeof(float) * TRI_COUNT(m_x) - m_x)
    	)> properties.totalGlobalMem)
    {
        printf("Input matrix size is larger than the available device memory!\n");
        printf("Please choose a smaller size matrix\n");

        // hipDeviceReset causes the driver to clean up all state. While
        // not mandatory in normal operation, it is good practice.  It is also
        // needed to ensure correct operation when the application is being
        // profiled. Calling hipDeviceReset causes all profile data to be
        // flushed before the application exits
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

	num_total_partitions = nBootstrap * nModelList * nKList * nReps;

    printf("\n\tStarting Ensemble Clustering on CUDA for total of %d partitions...\n", num_total_partitions);


	/*
	 * Transpose X as data stored in single dimensional array
	 * is row major not column
	 */

	x_transpose = (float*)malloc(mem_size);

	// Use x_transpose as temp buffer to convert double* to float*
//	std::copy(x_actual_data, x_actual_data + m_x* n_x, x_transpose);


	for (int i = 0; i < m_x; i++) {
		for (int j = 0; j < n_x; j++) {
			x_transpose[i*n_x+j] = x_actual_data[i + j * m_x];

		}
	}

	// Print PST to see
	printf("\n***************************************************************************\n");
	printf("\t\t\t\tPST\n");
	printf("***************************************************************************\n\n");

	printf("\nname: %s", name);
	printf("\nkList:\n");
	for (int j = 0; j < nKList; j++) {
		printf("\t%d\n", (int) kList[j]);
	}
	printf("\nModel List:\n");
	for (int j = 0; j < nModelList; j++) {
		printf("\t%s\n", modelList[j]);
	}

	printf("\noutlierCutoff: %f", outlierCutoff);
	printf("\noutlierMetric: %s", outlierMetric);

    printf("\n\nStarting Ensemble Clustering on CUDA...\n");
    double createen_time_spent = 0;
    // CUDA events
    hipEvent_t start, stop;
    // initialize events
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    /**
     * Optimization to reduce computation by reordering the models
     */
    char clust_algorithms[nModelList][MODEL_STR_LEN], clust_algorithms_distmetric[nModelList][MODEL_STR_LEN], clust_algorithms_centerfun[nModelList][MODEL_STR_LEN], clust_algorithms_linkcode[nModelList][MODEL_STR_LEN];
//    cout<<"\n\tBEFORE SORT"<<endl;
    for (int c_nModels = 0; c_nModels < nModelList; c_nModels++) {


    	distmetric = (char*) malloc(MODEL_STR_LEN * sizeof(char));
    	centerfun = (char*) malloc(MODEL_STR_LEN * sizeof(char));
    	linkcode = (char*) malloc(MODEL_STR_LEN * sizeof(char));
    	strcpy(distmetric, "\0");
    	strcpy(centerfun, "\0");
    	strcpy(linkcode, "\0");

    	// DO SPECTRAL
    	if(getModelType(SPECTRAL_SHRT, modelList[c_nModels])){
    		getDistMtrcnCntrFunBySpectral(SPECTRAL_SHRT, modelList[c_nModels], distmetric, centerfun);
    		//						cout<<distmetric << " : "<< centerfun<<endl;
    		strcpy(clust_algorithms[c_nModels], SPECTRAL_LNG);
    		strcpy(clust_algorithms_distmetric[c_nModels], distmetric);
    		strcpy(clust_algorithms_centerfun[c_nModels], centerfun);
    		strcpy(clust_algorithms_linkcode[c_nModels], linkcode);

    	}
    	// DO KMEANS
    	else if(getModelType(KMEANS_SHRT, modelList[c_nModels]))
    	{
    		getDistMtrcnCntrFunByKmeans(KMEANS_SHRT, modelList[c_nModels], distmetric, centerfun);
    		//						cout<<distmetric << " : "<< centerfun<<endl;
    		strcpy(clust_algorithms[c_nModels], KMEANS_LNG);
    		strcpy(clust_algorithms_distmetric[c_nModels], distmetric);
    		strcpy(clust_algorithms_centerfun[c_nModels], centerfun);
    		strcpy(clust_algorithms_linkcode[c_nModels], linkcode);

    	}
    	// DO KMEDOID
    	else if(getModelType(KMEDOIDS_SHRT, modelList[c_nModels]))
    	{
    		getDistMtrcnCntrFunByKmedoid(KMEDOIDS_SHRT, modelList[c_nModels], distmetric, centerfun);
    		//						cout<<distmetric << " : "<< centerfun<<endl;
    		strcpy(clust_algorithms[c_nModels], KMEDOIDS_LNG);
    		strcpy(clust_algorithms_distmetric[c_nModels], distmetric);
    		strcpy(clust_algorithms_centerfun[c_nModels], centerfun);
    		strcpy(clust_algorithms_linkcode[c_nModels], linkcode);


    	}
    	// DO GMM
    	else if(getModelType(GMM_SHRT, modelList[c_nModels]))
    	{
    		strcpy(clust_algorithms[c_nModels], GMM_LNG);
    		strcpy(clust_algorithms_distmetric[c_nModels], distmetric);
    		strcpy(clust_algorithms_centerfun[c_nModels], centerfun);
    		strcpy(clust_algorithms_linkcode[c_nModels], linkcode);

    	}
    	// DO KMEDOID
    	else if(getModelType(AGGLO_SHRT, modelList[c_nModels]))
    	{
    		getDistMtrcnCntrFunByAgglo(AGGLO_SHRT, modelList[c_nModels], distmetric, centerfun, linkcode);
    		strcpy(clust_algorithms[c_nModels], AGGLO_LNG);
    		strcpy(clust_algorithms_distmetric[c_nModels], distmetric);
    		strcpy(clust_algorithms_centerfun[c_nModels], centerfun);
    		strcpy(clust_algorithms_linkcode[c_nModels], linkcode);

    	}
//    	cout<<"\t"<<clust_algorithms[c_nModels]<< " : "<< clust_algorithms_distmetric[c_nModels]<< " : "<< clust_algorithms_centerfun[c_nModels]<< " : "<< clust_algorithms_linkcode[c_nModels] <<endl;
		free(distmetric);
		free(centerfun);
		free(linkcode);
    }

    sort_models(clust_algorithms_distmetric, nModelList, clust_algorithms, clust_algorithms_centerfun, clust_algorithms_linkcode, modelList);

//    cout<<"\n\tAFTER SORT"<<endl;
//    for (int c_nModels = 0; c_nModels < nModelList; c_nModels++) {
//    	cout<<"\t"<<clust_algorithms[c_nModels]<< " : "<< clust_algorithms_distmetric[c_nModels]<< " : "<< clust_algorithms_centerfun[c_nModels]<< " : "<< clust_algorithms_linkcode[c_nModels] <<endl;
//    }

    /**
     * check to make sure max k is not too big for data set
     */

    clock_t begin = clock();

    double max_value = thrust::reduce(thrust::host,	kList, kList + nKList, -1, thrust::maximum<double>());

    if( max_value > m_x/3){
    	cout<<"\nThe largest value of k in the kList is too big for the data set. The largest k should be less than 1/3 of nSamples.\n"<<endl;
    	return (EXIT_FAILURE);
    }

    /**
     * Pre-Process data
     */
    printf("\n\nStarting pre-processing...\n");
    bool *outlierMask;
    int* outlierIdxs_data;
    int* outlierIdxs_size;
    float* outlierZ_data;
    int* outlierZ_size;

    outlierCutoff = 1;
    int nrows_no_outliers = 0;
    float *x_outlier_removed = cluster_data_outliers(m_x,
    						 n_x,
    						 x_transpose,
    						 outlierMetric,
    						 FLT_MAX/*outlierCutoff*/,
    						 &nrows_no_outliers);

    int nrows = nrows_no_outliers;
    int ncols = n_x;

    printf("\n\nDone pre-processing...\n");
    printf("\n\n%d objects were removed from original dataset...\n",m_x - nrows);
    clock_t end = clock();

    double preprocessing_time_spent = (double)(end - begin)/(double)CLOCKS_PER_SEC;
    printf("\n***************************************************************************\n");
    printf("Time spent in pre-processing for %d objects is %f\n", m_x, preprocessing_time_spent);
    printf("***************************************************************************\n\n");

    printf("\n\nGenerating boot indices...\n");

    mxArray *bootIdxsm = NULL;
    bootIdxsm = mxCreateNumericMatrix(nrows, nBootstrap, mxINT32_CLASS, mxREAL);
    bootIdxs = (int*)mxGetPr(bootIdxsm);
    int *bootIdxs2 = (int*)malloc(nrows * nBootstrap * sizeof(int));

    for(int row=0; row < nrows; row++){
    	for(int col=0; col < nBootstrap; col++){
    		bootIdxs2[row * nBootstrap + col] = (rand()%(nrows-0))+0;;
    		bootIdxs[row + col * nrows] = bootIdxs2[row * nBootstrap + col];
    	}
    }

	/*Do the clustering*/
	printf("\n***************************************************************************\n");
	printf("\t\t\tGenerating cluster ensemble\n");
	printf("***************************************************************************\n\n");

	data_partition_indices = (int*) malloc(num_total_partitions * nrows * sizeof(int));

	printf("\n\tRunning Ensemble with %d Bootstrap...\n\n", nBootstrap);

	//Setup the output .mat file structure.
	int total_models = 0;
	int index, nsubs=5;
	const char *field_names_e[] = {"name", "description","parameters","inputData","bootIndices", "partitions", "ensolution"};
	int num_fields_e = 7; // Basically length of [] field_names_e
	const char *field_names_partitions[] = {"model", "boot","k","rep","indices"};
	int num_fields_partitions = 5; // Basically length of [] field_names_partitions
	size_t subs[2];
	mxArray *partitions_array_ptr;
//	output_cell_array_ptr = mxCreateCellMatrix(nBootstrap*nModelList*nKList*nReps , nsubs);

	mwSize dims_partitions[2] = {1, nBootstrap*nModelList*nKList*nReps };
	mwSize dims_e[2] = {1, 1 };
	/* Create a 1-by-n array of structs. */
	partitions_array_ptr = mxCreateStructArray(2, dims_partitions, num_fields_partitions, field_names_partitions);

	output_cell_array_ptr = mxCreateStructArray(2, dims_e, num_fields_e, field_names_e);

	// Set partitions struct in output
	mxSetFieldByNumber(output_cell_array_ptr,0, mxGetFieldNumber(output_cell_array_ptr,field_names_e[0]), mxCreateString(en_name));
	mxSetFieldByNumber(output_cell_array_ptr,0, mxGetFieldNumber(output_cell_array_ptr,field_names_e[1]), mxCreateString(en_desc));
	mxSetFieldByNumber(output_cell_array_ptr,0, mxGetFieldNumber(output_cell_array_ptr,field_names_e[2]), aPST);
	mxSetFieldByNumber(output_cell_array_ptr,0, mxGetFieldNumber(output_cell_array_ptr,field_names_e[4]), bootIdxsm);

	mxArray *Xm = NULL;

	// copy data to E (output struct).
	Xm = mxCreateDoubleMatrix (nrows, ncols, mxREAL);
	double *X_p;
	X_p = mxGetPr(Xm);
	for (int i = 0; i < nrows; i++){
#pragma unroll
		for (int j = 0; j < ncols; j++){
			X_p[i + j * nrows] = x_outlier_removed[i * ncols + j];
		}
	}
	mxSetFieldByNumber(output_cell_array_ptr,0, mxGetFieldNumber(output_cell_array_ptr,field_names_e[3]), Xm);

	// Allocate memory for distance matrix

	float *distmatrix_device, *distmatrix_host;
	CUDA_CHECK_RETURN(hipMalloc(&distmatrix_device, TRI_COUNT(nrows)*sizeof(float)));
	assert(distmatrix_device != NULL);
	distmatrix_host = (float*) malloc(TRI_COUNT(nrows)*sizeof(float));
	assert(distmatrix_host != NULL);
	int *tri_idxs_device;
	int *tri_idxs_host;
	float* weight_host;
	float* weight_device;
	size_t memWtSz = 0, memIdxSz=0;
	int *maxGridSize;
	int *maxThreadsPerBlock;
	Node* tree = NULL;

	CUDA_CHECK_RETURN(hipMalloc(&maxGridSize, sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc(&maxThreadsPerBlock, sizeof(int)));

	CUDA_CHECK_RETURN(hipMemcpy(maxGridSize, &properties.maxGridSize[1], sizeof(int), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(maxThreadsPerBlock, &properties.maxThreadsPerBlock, sizeof(int), hipMemcpyHostToDevice));



	memWtSz = ncols * sizeof(float);

	CUDA_CHECK_RETURN(hipMalloc(&weight_device, memWtSz));
	assert(weight_device != NULL);

	weight_host = (float*) malloc(memWtSz);
	assert(weight_host != NULL);

	memIdxSz = TRI_COUNT(nrows)*sizeof(int);

	CUDA_CHECK_RETURN(hipMalloc(&tri_idxs_device, memIdxSz));

	assert(tri_idxs_device != NULL);

	tri_idxs_host = (int*) malloc(memIdxSz);

	for (int i = 0; i < ncols; i++)
		weight_host[i] = 1.0;
	CUDA_CHECK_RETURN(hipMemcpy(weight_device, weight_host, memWtSz, hipMemcpyHostToDevice));

	int jCount = 0;
	for (int i = 1; i < nrows; i++){
		for (int j = 0; j < i; j++){
			tri_idxs_host[jCount] = TRI_COUNT(i)+j;
			jCount++;
		}
	}
	CUDA_CHECK_RETURN(hipMemcpy(tri_idxs_device, tri_idxs_host, memIdxSz, hipMemcpyHostToDevice));

	hipResourceDesc resDescWt;
	memset(&resDescWt, 0, sizeof(resDescWt));
	resDescWt.resType = hipResourceTypeLinear;
	resDescWt.res.linear.devPtr = weight_device;
	resDescWt.res.linear.desc.f = hipChannelFormatKindFloat;
	resDescWt.res.linear.desc.x = 32; // bits per channel
	resDescWt.res.linear.sizeInBytes = memWtSz;

	hipResourceDesc resDescIdx;
	memset(&resDescIdx, 0, sizeof(resDescIdx));
	resDescIdx.resType = hipResourceTypeLinear;
	resDescIdx.res.linear.devPtr = tri_idxs_device;
	resDescIdx.res.linear.desc.f = hipChannelFormatKindSigned;
	resDescIdx.res.linear.desc.x = 32; // bits per channel
	resDescIdx.res.linear.sizeInBytes = memIdxSz;

	hipTextureDesc texDescWt;
	memset(&texDescWt, 0, sizeof(texDescWt));
	texDescWt.readMode = hipReadModeElementType;

	hipTextureDesc texDescIdx;
	memset(&texDescIdx, 0, sizeof(texDescIdx));
	texDescIdx.readMode = hipReadModeElementType;

	hipTextureObject_t texWt = 0;
	hipCreateTextureObject(&texWt, &resDescWt, &texDescWt, NULL);
	if (hipSuccess != (err = hipGetLastError()))
	{
		printf("\n8. @texWt error: %s\n", hipGetErrorString(err));
		return(EXIT_FAILURE);
	}

	hipTextureObject_t texIdx = 0;
	hipCreateTextureObject(&texIdx, &resDescIdx, &texDescIdx, NULL);
	if (hipSuccess != (err = hipGetLastError())) {
		printf("\n8. @texIdx error: %s\n", hipGetErrorString(err));
		return(EXIT_FAILURE);
	}

	/*
	 * Below loops go through
	 * 1. nBootstrap data samples
	 * 2. nModelList e.g. 'kmeanseuc','kmeanscit','kmeanscor','kmeanscos',...
	 * 					  'spectraleuc','spectralcit','spectralcor','spectralcos',...
	 * 					  'aggeucwar','aggeucavg','aggeuccom',...
	 * 					  'aggcitavg','aggcitcom',...
	 * 					  'aggcoravg','aggcorcom',...
	 * 					  'aggcosavg','aggcoscom'
	 * 3. nKList: Number of clusters to be found in data for combinations of bootstrap data and model  e.g. 1, 2, 3..5,10
	 * 4. nReps: Repeat the model (combination of bootstrap data sample, model, #cluster) to avoid bias.
	 */
	for (int c_nBootstrap = 0; c_nBootstrap < nBootstrap; c_nBootstrap++) {

		/*
		 * Variables to keep track of memory used on device.
		 * Depending on the current memory usage we need to restrict
		 * number of threads.
		 */
		int* mask;

		float* data1d_device;
		float* data1d_host;

		size_t memDataSz = 0;

		memDataSz = nrows * ncols * sizeof(float);
		CUDA_CHECK_RETURN(hipMalloc(&data1d_device, memDataSz));
		assert(data1d_device!=NULL);
		data1d_host = (float*) malloc(nrows*ncols*sizeof(float));
		assert(data1d_host!=NULL);


		/**
		 * Generate bootstrap data and write it
		 * to a temporary file to use by clustering
		 * algorithms.
		 * Create bootstrap data sets...
		 */
		int *b = (int*) malloc(nrows * sizeof(int));

		for (int j = 0; j < nrows; j++) {
			memcpy(	&data1d_host[j * ncols], &x_outlier_removed[bootIdxs2[j * nBootstrap + c_nBootstrap] * ncols], ncols * sizeof(float));
			b[j] = bootIdxs2[j * nBootstrap + c_nBootstrap];
		}
		CUDA_CHECK_RETURN(hipMemcpy(data1d_device, data1d_host, memDataSz, hipMemcpyHostToDevice));

		//TODO: Debug code remove later!
//		printf("\n-------------------------------------------------------------------------------------- \n\n");
//		for (int m = 0; m < nrows; m++) {
//			for (int n = 0; n < ncols; n++) {
//
////				data1d_managed[n + m * ncols] = x_outlier_removed[bootIdxs2[m * nBootstrap + c_nBootstrap] * ncols+n];
////				printf("%.3f ", data1d_managed[n + m * ncols]);
//				printf("%.3f ", x_outlier_removed[(bootIdxs2[m * nBootstrap + c_nBootstrap] * ncols)+n]);
////				printf("%d ", bootIdxs2[m * nBootstrap + c_nBootstrap]>=nrows-1?1:0);
//			}
//			b[m] = bootIdxs2[m * nBootstrap + c_nBootstrap];
//			printf(" \n");
//		}

//		printf("\n NO PROBLEM TILL HERE \n");


		// bind texture to buffer
		// create texture object
		hipResourceDesc resDescData;
		memset(&resDescData, 0, sizeof(resDescData));
		resDescData.resType = hipResourceTypeLinear;
		resDescData.res.linear.devPtr = data1d_device;
		resDescData.res.linear.desc.f = hipChannelFormatKindFloat;
		resDescData.res.linear.desc.x = 32; // bits per channel
		resDescData.res.linear.sizeInBytes = memDataSz;

		hipTextureDesc texDescData;
		memset(&texDescData, 0, sizeof(texDescData));
		texDescData.readMode = hipReadModeElementType;


		// create texture object: we only have to do this once!
		hipTextureObject_t texData = 0;
		hipCreateTextureObject(&texData, &resDescData, &texDescData, NULL);
		if (hipSuccess != (err = hipGetLastError()))
		{
			printf("\n6. @texData error: %s\n",	hipGetErrorString(err));
			return(EXIT_FAILURE);
		}


		distmetric = (char*) malloc(MODEL_STR_LEN * sizeof(char));
		centerfun = (char*) malloc(MODEL_STR_LEN * sizeof(char));
		linkcode = (char*) malloc(MODEL_STR_LEN * sizeof(char));

		for (int c_nModels = 0; c_nModels < nModelList; c_nModels++) {

			if(strcmp(distmetric, clust_algorithms_distmetric[c_nModels]) != 0){
//				cout<<"\n\t Distance metric changed to " <<clust_algorithms_distmetric[c_nModels]<<endl;

				int maxThreads = properties.maxThreadsPerBlock;  // number of threads per block
				int threads = (TRI_COUNT(nrows)+nrows < maxThreads) ? nextPow2Host((TRI_COUNT(nrows)+nrows)) : maxThreads;
				int gridSize = ( TRI_COUNT(nrows) + threads - 1) / threads < properties.maxGridSize[1]? (TRI_COUNT(nrows) + threads - 1)/ threads : properties.maxGridSize[1];
				char dist = dist_str2c(clust_algorithms_distmetric[c_nModels]);
				//	printf("\nCalling distance matrix calculation kernel with gridSize = %d Threads = %d for TRI_COUNT(%d)=%d", gridSize, threads, nrows, TRI_COUNT(nrows));
				if(threads > 0 && gridSize > 0)
					distancematrix<<< gridSize, threads>>>(nrows, ncols , dist, distmatrix_device, texData, texWt,texIdx, maxGridSize, maxThreadsPerBlock);

				// wait for child to complete
				if (hipSuccess != hipDeviceSynchronize()) {
					return (EXIT_FAILURE);
				}

				CUDA_CHECK_RETURN(hipMemcpy(distmatrix_host, distmatrix_device, TRI_COUNT(nrows)*sizeof(float), hipMemcpyDeviceToHost));

				if (hipSuccess != (err=hipGetLastError())) {
					printf("\n9. @distancematrix copy error: %s\n", hipGetErrorString(err));
					return (EXIT_FAILURE);
				}

			}
			if(strcmp(linkcode, clust_algorithms_linkcode[c_nModels]) != 0){
//				cout<<"\n\t Linkage changed to " <<clust_algorithms_linkcode[c_nModels]<<endl;
				if(tree != NULL) free(tree);
				tree = NULL;
			}
//			cout<<"\t"<<clust_algorithms[c_nModels]<< " : "<< clust_algorithms_distmetric[c_nModels]<< " : "<< clust_algorithms_centerfun[c_nModels]<< " : "<< clust_algorithms_linkcode[c_nModels] <<endl;

			strcpy(distmetric, clust_algorithms_distmetric[c_nModels]);
			strcpy(centerfun, clust_algorithms_centerfun[c_nModels]);
			strcpy(linkcode, clust_algorithms_linkcode[c_nModels]);


			for (int c_nKList = 0; c_nKList < nKList; c_nKList++) {
				for (int c_nReps = 1; c_nReps < nReps+1; c_nReps++) {
					mxArray *idxs_ptr = mxCreateNumericMatrix(1, nrows, mxINT32_CLASS, mxREAL);
					int *mxIdxs = (int*)mxGetData(idxs_ptr);

					int *idxs = (int*)malloc(nrows * sizeof(int));
					assert(idxs != NULL);
					assert(mxIdxs != NULL);

					clock_t begin = clock();

				     // take measurements for loop over kernel launches
//				     checkCudaErrors(hipEventRecord(start, 0));

					// DO SPECTRAL
					if(getModelType(SPECTRAL_SHRT, clust_algorithms[c_nModels])){

//						getDistMtrcnCntrFunBySpectral(SPECTRAL_SHRT, modelList[c_nModels], distmetric, centerfun);
//						cout<<distmetric << " : "<< centerfun<<endl;
//						mex2cu_spectral_adapter((int)kList[c_nKList], nrows, ncols, NULL, clust_algorithms_distmetric[c_nModels], idxs, texData, texWt, texIdx);
						// Changed to optimize recomputation of inter object distances...
						mex2cu_spectral_adapter((int)kList[c_nKList], nrows, ncols, distmatrix_host, NULL, "euc", mxIdxs);
					}
					// DO KMEANS
					else if(getModelType(KMEANS_SHRT, clust_algorithms[c_nModels]))
					{
						float threshold = 0.001;
						int loop_iterations;

//						getDistMtrcnCntrFunByKmeans(KMEANS_SHRT, modelList[c_nModels], distmetric, centerfun);
//						cout<<distmetric << " : "<< centerfun<<endl;
						mex2cu_kmeans_adapter((int)kList[c_nKList], nrows, ncols, data1d_host, NULL, clust_algorithms_distmetric[c_nModels], threshold, &loop_iterations, idxs, texData, texWt);
					}
					// DO KMEDOID
					else if(getModelType(KMEDOIDS_SHRT, clust_algorithms[c_nModels]))
					{
						float threshold = 0.001;
						int loop_iterations;

//						getDistMtrcnCntrFunByKmedoid(KMEDOIDS_SHRT, modelList[c_nModels], distmetric, centerfun);
//						cout<<distmetric << " : "<< centerfun<<endl;
						mex2cu_kmedians_adapter((int)kList[c_nKList], nrows, ncols, data1d_host, NULL, clust_algorithms_distmetric[c_nModels], threshold, &loop_iterations, idxs, texData, texWt);
					}
					// DO KMEDOID
					else if(getModelType(GMM_SHRT, clust_algorithms[c_nModels]))
					{
						mex2cu_gmm_adapter((int)kList[c_nKList], nrows, ncols, data1d_host, NULL, NULL, idxs, texData, texWt);
					}
					// DO KMEDOID
					else if(getModelType(AGGLO_SHRT, clust_algorithms[c_nModels]))
					{
//						getDistMtrcnCntrFunByAgglo(AGGLO_SHRT, modelList[c_nModels], distmetric, centerfun, linkcode);
//						cout<<clust_algorithms_distmetric[c_nModels] << " : "<< clust_algorithms_linkcode[c_nModels]<<endl;
						tree = mex2cu_agglomerative_adapter((int)kList[c_nKList], nrows, ncols, data1d_host, weight_host, clust_algorithms_linkcode[c_nModels], clust_algorithms_distmetric[c_nModels], idxs, distmatrix_host, tree, texData, texWt, texIdx);
					}

//			        checkCudaErrors(hipEventRecord(stop, 0));
//			        checkCudaErrors(hipEventSynchronize(stop));
//					float kernelTime;
//					checkCudaErrors(hipEventElapsedTime(&kernelTime, start, stop));

					clock_t end = clock();
					double locat_time_spent = (double)(end - begin)/(double)CLOCKS_PER_SEC;
					createen_time_spent += locat_time_spent;
//					time_spent += kernelTime;
#ifdef MEASURE_TIME
					printf("\tTime spent in %+15s (model), %4d (bootstrap), %3d (k), %2d-Rep  is: %3.4f\n", modelList[c_nModels], c_nBootstrap, (int)kList[c_nKList], c_nReps, locat_time_spent);
#endif
//					for(int i = 0; i < nrows; i++){
//						printf("%d ", idxs[i]);
//					}
//					printf("\n");

					/**
					 * cluster indices for boot sample need to be converted to
					 * cluster indices for original data samples
					 * NB:  do this BEFORE computing the centers...so centers
					 * computed only based on original data, not boostrap data
					 */
					cluster_util_bootpartition2partition(nrows, b, idxs, mxIdxs);
					/**
					 * Copy data partition indices into a bigger storage matrix
					 * that will be used later to extract ensemble solution later.
					 */
					memcpy(&data_partition_indices[total_models * nrows], mxIdxs, nrows * sizeof(int));

					/* Place the Model string array into cell element (total_models,0). */
					mxSetFieldByNumber(partitions_array_ptr,total_models, mxGetFieldNumber(partitions_array_ptr,field_names_partitions[0]),mxCreateString(modelList[c_nModels]));

					/* Place the Bootstrap number into cell element (total_models,1). */
					mxSetFieldByNumber(partitions_array_ptr,total_models, mxGetFieldNumber(partitions_array_ptr,field_names_partitions[1]),mxCreateDoubleScalar(c_nBootstrap));
					/* Place the #cluster (k) into cell element (total_models,2). */
					mxSetFieldByNumber(partitions_array_ptr,total_models, mxGetFieldNumber(partitions_array_ptr,field_names_partitions[2]), mxCreateDoubleScalar(kList[c_nKList]));

					/* Place the #repition into cell element (total_models,3). */
					mxSetFieldByNumber(partitions_array_ptr,total_models, mxGetFieldNumber(partitions_array_ptr,field_names_partitions[3]), mxCreateDoubleScalar(c_nReps));

					/* Place the idx into cell element (total_models,4). */
					mxSetFieldByNumber(partitions_array_ptr,total_models, mxGetFieldNumber(partitions_array_ptr,field_names_partitions[4]), idxs_ptr);
					//free(idxs);


					total_models++;

					free(idxs);
				}

			}
		}


		free(distmetric);
		free(centerfun);
		free(linkcode);

		// destroy texture object
		hipDestroyTextureObject(texData);

		CUDA_CHECK_RETURN(hipFree(data1d_device));

		free(data1d_host);
		free(b);
	}
	if(tree != NULL) free(tree);
	tree = NULL;
	hipDestroyTextureObject(texWt);
	hipDestroyTextureObject(texIdx);
	CUDA_CHECK_RETURN(hipFree(tri_idxs_device));
	CUDA_CHECK_RETURN(hipFree(weight_device));
	CUDA_CHECK_RETURN(hipFree(distmatrix_device));
	free(distmatrix_host);
	free(tri_idxs_host);
	free(weight_host);
	CUDA_CHECK_RETURN(hipFree(maxGridSize));
	CUDA_CHECK_RETURN(hipFree(maxThreadsPerBlock));


	// Set partitions struct in output
	mxSetFieldByNumber(output_cell_array_ptr,0, mxGetFieldNumber(output_cell_array_ptr,field_names_e[5]), partitions_array_ptr);

	// Print a message describing what the sample does.
	printf("\n***************************************************************************\n");
	printf("Time spent in Create Ensemble Clusters for %d objects is %f\n", m_x, createen_time_spent);
	//    printf("=%d blocks are launched!!! (%d from the GPU)\n", sum, sum-2);
	printf("***************************************************************************\n\n");

	double extract_time_spent = 0;
	/*----------------------------------START OF ENSEMBLE EXTRACTIN CODE----------------------*/
#ifdef EXTRACT_ENSEMBLE_SOLUTION
	printf("\n***************************************************************************\n");
	printf("\t\t\tExtracting Solution from ensemble\n");
	printf("***************************************************************************\n\n");

	pst_extract = mxGetField(aPST, 0, MEX_STRUCT_PST_FIELD_EXTRACT);

	tmp = mxGetField(pst_extract, 0, MEX_STRUCT_PST_FIELD_EXTRACT_KLIST);
	int extract_nKList = mxGetN(tmp);
	double *extract_kList = mxGetPr(tmp);

	const char *field_names_ensolution[] = {"dissimMatrix","coassocSoln"};
	int num_fields_ensolution = 2; // Basically length of [] field_names_partitions
	mxArray *ensolution_array_ptr;
	mwSize dims_ensolution[2] = {1, 1};

	const char *field_names_coassocSoln[] = {"k", "centers","clusterN","indices","clusterSSW"};
	int num_fields_coassocSoln = 5; // Basically length of [] field_names_partitions

	mxArray *coassocSoln_array_ptr;

	mwSize dims_coassocSoln[2] = {1, extract_nKList};

	/* Create a 1-by-n array of structs. */
	coassocSoln_array_ptr = mxCreateStructArray(2, dims_coassocSoln, num_fields_coassocSoln, field_names_coassocSoln);

	/* Create a 1-by-n array of structs. */
	ensolution_array_ptr = mxCreateStructArray(2, dims_ensolution, num_fields_ensolution, field_names_ensolution);

	mxArray *dissimMatrix_ptr = mxCreateNumericMatrix(nrows, nrows, mxDOUBLE_CLASS, mxREAL);
	double *mxdissimMatrix = (double*)mxGetData(dissimMatrix_ptr);

	float *A, *N;
	A = (float*) malloc(nrows * nrows * sizeof(float));
	N = (float*) malloc(nrows * nrows * sizeof(float));

	printf("\n\tConstructing co-association matrix from %d partitions...", num_total_partitions);
	cluster_ensemble2cam(data_partition_indices,  A, N, nrows, num_total_partitions);

	// TODO: SAVE A TO OUTPUT STRUCTURE

	for(int i =0; i < nrows; i++){
		for(int j=0; j< nrows; j++){
			mxdissimMatrix[i + j * nrows] = A[i * nrows + j];
			//			printf("%f ", A[i * nrows + j]);
		}
		//		printf("\n");
	}

	// save dissimMatrix
	mxSetFieldByNumber(ensolution_array_ptr,0, mxGetFieldNumber(ensolution_array_ptr,field_names_ensolution[0]), dissimMatrix_ptr);

	/**
	 * use spectral clustering to get solutions for each k in kList
	 */
	printf("\n\tClustering the co-association matrix...");
	begin = clock();
	for (int c_nKList = 0; c_nKList < extract_nKList; c_nKList++) {

		mxArray *idxs_ptr = mxCreateNumericMatrix(1, nrows, mxINT32_CLASS, mxREAL);
		int *mxIdxs = (int*)mxGetData(idxs_ptr);

		// use spectral clustering to get solutions for each k in kList
		mex2cu_spectral_adapter((int)extract_kList[c_nKList], nrows, nrows, A, NULL, "euc", mxIdxs);

		/**
		 * Computes cluster centers based on indices
		 */
		int nClusters = unique_length(mxIdxs, nrows);
		float *M = (float*) malloc(nClusters * ncols * sizeof(float));

		cluster_util_indices2centers(x_outlier_removed, nrows, ncols, mxIdxs, "mean", nClusters, M);

		mxArray *M_ptr = mxCreateNumericMatrix(nClusters, ncols, mxDOUBLE_CLASS, mxREAL);
		double *M_data = (double*)mxGetData(M_ptr);

		for(int i =0; i < nClusters; i++){
			for(int j=0; j< ncols; j++){
				M_data[i + j * nClusters] = M[i * ncols + j];
			}
		}
		free(M);

		/**
		 * Count how many objects were assigned to each labels
		 */
		mxArray *valueList_ptr = mxCreateNumericMatrix(1, nClusters, mxINT32_CLASS, mxREAL);
		int *valueList = (int*)mxGetData(valueList_ptr);
		indices_count(mxIdxs, nrows, valueList, nClusters);

		/**
		 *  Compute the sum squared error within, for each cluster
		 */
		mxArray *ssw_ptr = mxCreateNumericMatrix(1, nClusters, mxDOUBLE_CLASS, mxREAL);
		double *ssw_data = (double*) mxGetData(ssw_ptr);

		cluster_util_ssw(x_outlier_removed, nrows, ncols, mxIdxs, ssw_data, nClusters);

		/* Place the #cluster (k) into cell element (c_nKList,0). */
		mxSetFieldByNumber(coassocSoln_array_ptr,c_nKList, mxGetFieldNumber(coassocSoln_array_ptr,field_names_coassocSoln[0]), mxCreateDoubleScalar(extract_kList[c_nKList]));

		/* Place the centers matrix into cell element (c_nKList,1). */
		mxSetFieldByNumber(coassocSoln_array_ptr,c_nKList, mxGetFieldNumber(coassocSoln_array_ptr,field_names_coassocSoln[1]), M_ptr);

		/* Place the number objects were assigned to each labels into cell element (c_nKList,2). */
		mxSetFieldByNumber(coassocSoln_array_ptr,c_nKList, mxGetFieldNumber(coassocSoln_array_ptr,field_names_coassocSoln[2]), valueList_ptr);

		/* Place the idx into cell element (c_nKList,3). */
		mxSetFieldByNumber(coassocSoln_array_ptr,c_nKList, mxGetFieldNumber(coassocSoln_array_ptr,field_names_coassocSoln[3]), idxs_ptr);

		/* Place the ssw into cell element (c_nKList,4). */
		mxSetFieldByNumber(coassocSoln_array_ptr,c_nKList, mxGetFieldNumber(coassocSoln_array_ptr,field_names_coassocSoln[4]), ssw_ptr);

	}

	end = clock();
	extract_time_spent = (double)(end - begin)/(double)CLOCKS_PER_SEC;

	printf("\n\n***************************************************************************\n");
	printf("Time spent in Extracting Solution from ensemble for %d objects is %f\n", m_x, extract_time_spent);
	printf("***************************************************************************\n\n");

	// Set coassocSoln struct in ensolution struct
	mxSetFieldByNumber(ensolution_array_ptr,0, mxGetFieldNumber(ensolution_array_ptr,field_names_ensolution[1]), coassocSoln_array_ptr);
	// Set ensolution struct in output
	mxSetFieldByNumber(output_cell_array_ptr,0, mxGetFieldNumber(output_cell_array_ptr,field_names_e[6]), ensolution_array_ptr);
	/*-----------------------------------END OF ENSEMBLE EXTRACTIN CODE-----------------------*/
#endif


	// TODO: might want to create a seperate method for output writing (but I am lazy for now)
	remove(output_file);
	MATFile *pmat = matOpen(output_file, "w");

	if (pmat == NULL) {
		printf("Error reopening file %s\n", output_file);
		return(EXIT_FAILURE);
	}
	int status = matPutVariable(pmat, "E", output_cell_array_ptr);
	if (status != 0) {
		printf("%s :  Error using matPutVariable on line %d\n", __FILE__, __LINE__);
		return(EXIT_FAILURE);
	}

	if (matClose(pmat) != 0) {
		printf("Error closing file %s\n",output_file);
		return(EXIT_FAILURE);
	}

    mxDestroyArray(aPST);
    mxDestroyArray(aX);


    if (matClose(mfPtr) != 0) {
        printf("Error closing file %s\n", argv[1]);
        return(EXIT_FAILURE);
    }
    if (matClose(mfPtr2) != 0) {
    	printf("Error closing file %s\n", argv[2]);
    	return(EXIT_FAILURE);
    }


    cout<<endl<<"\n\t...ensemble generation complete :)\n"<<endl;
    printf("\n\n***************************************************************************\n");
    printf("Time spent in Ensemble Clustering for %d objects is %f\n", m_x, preprocessing_time_spent + createen_time_spent + extract_time_spent);
    printf("***************************************************************************\n\n");
//    CUDA_CHECK_RETURN(hipDeviceReset());
    free(x_transpose);
	free(modelList);
    free(data_partition_indices);
    free(bootIdxs2);
    return(EXIT_SUCCESS);
}

void testSpectral(){
    // Declare Variables
	string line;
	int testcase = 1;

	int nrows = 0, ncols = 0;
	int    *idxs;    /* [numObjs] */
	float *data1d;
	int numClusters = 0;

//	for (int row = 0; row < nrows; row++){
//		data1d[row] = (double*)malloc(nrows* sizeof(double));
//		for (int col = 0; col < ncols; col++){
//
//			data1d[row][col] = ((float)rand() / (RAND_MAX)) + 1;
//
////			cout << " "<<data1d[row][col];
//
//		}
////		cout<<endl;
//	}

	switch(testcase){
	case 1:{
		nrows = 10;
		ncols = 3;
		numClusters = 3;

		ifstream in("dataset/data10x3.txt");

		/* start the timer for the core computation -----------------------------*/
		/* membership: the cluster id for each data object */
		idxs = (int*)malloc(nrows * sizeof(int));
		assert(idxs != NULL);
		data1d = (float*)malloc(nrows*ncols * sizeof(float));
		assert(data1d != NULL);
		if (!in) {
			cout << "Cannot open file.\n";
			exit(EXIT_SUCCESS);
		}

		for(int i = 0; i < nrows; i++)
		{

			for(int j = 0; j < ncols; j++)
			{
				in >> data1d[i*ncols + j];

				       		         printf("%f ",data1d[i*ncols + j]);
			}
			       		        printf("\n");

		}
		in.close();

	}
	break;
	case 2:{
		nrows = 10;
		ncols = 3;
		numClusters = 3;

		ifstream in("dataset/data10x3-2.txt");

		/* start the timer for the core computation -----------------------------*/
		/* membership: the cluster id for each data object */
		idxs = (int*)malloc(nrows * sizeof(int));
		assert(idxs != NULL);
		data1d = (float*)malloc(nrows*ncols * sizeof(float));
		assert(data1d != NULL);
		if (!in) {
			cout << "Cannot open file.\n";
			exit(EXIT_SUCCESS);
		}

		for(int i = 0; i < nrows; i++)
		{

			for(int j = 0; j < ncols; j++)
			{
				in >> data1d[i*ncols + j];

				//       		         printf(" %f",data1d[i][j]);
			}
			//       		        printf("\n");

		}
		in.close();

	}
	break;
	case 3:{
			nrows = 10;
			ncols = 20;
			numClusters = 4;

			ifstream in("dataset/data10x20.txt");

			/* start the timer for the core computation -----------------------------*/
			/* membership: the cluster id for each data object */
			idxs = (int*)malloc(nrows * sizeof(int));
			assert(idxs != NULL);
			data1d = (float*)malloc(nrows*ncols * sizeof(float));
			assert(data1d != NULL);
			if (!in) {
				cout << "Cannot open file.\n";
				exit(EXIT_SUCCESS);
			}

			for(int i = 0; i < nrows; i++)
			{

				for(int j = 0; j < ncols; j++)
				{
					in >> data1d[i*ncols + j];

					       		         printf("%f ",data1d[i*ncols + j]);
				}
				       		        printf("\n");

			}
			in.close();

		}
		break;

	case 4:{
		nrows = 2048;
		ncols = 20;
		ifstream in("dataset/data2048x20.txt");

		/* start the timer for the core computation -----------------------------*/
		/* membership: the cluster id for each data object */
		idxs = (int*)malloc(nrows * sizeof(int));
		assert(idxs != NULL);
		data1d = (float*)malloc(nrows*ncols * sizeof(float));

		if (!in) {
			cout << "Cannot open file.\n";
			exit(EXIT_SUCCESS);
		}

		for(int i = 0; i < nrows; i++)
		{

			for(int j = 0; j < ncols; j++)
			{
				in >> data1d[i*ncols + j];

				//	         printf(" %f",data1d[i][j]);
			}
			//	        printf("\n");

		}
		in.close();
	}
	break;

	case 5:{
		nrows = 4096;
		ncols = 20;
		ifstream in("dataset/data4096x20.txt");

		/* start the timer for the core computation -----------------------------*/
		/* membership: the cluster id for each data object */
		idxs = (int*)malloc(nrows * sizeof(int));
		assert(idxs != NULL);
		data1d = (float*)malloc(nrows*ncols * sizeof(float));

		if (!in) {
			cout << "Cannot open file.\n";
			exit(EXIT_SUCCESS);
		}

		for(int i = 0; i < nrows; i++)
		{

			for(int j = 0; j < ncols; j++)
			{
				in >> data1d[i*ncols + j];

				//	         printf(" %f",data1d[i][j]);
			}
			//	        printf("\n");

		}
		in.close();

	}
	break;
	}


	clock_t begin = clock();


//	mex2cuw_spectral_adapter(numClusters, nrows, ncols, data1d, NULL, "euc", idxs);

	clock_t end = clock();
	double time_spent = (double)(end - begin)/(double)CLOCKS_PER_SEC;


//    // Print a message describing what the sample does.
    printf("\n***************************************************************************\n");
	printf("\nTime spent to cluster %d objects using Spectral Clustering is %f\n", nrows, time_spent);
//    printf("=%d blocks are launched!!! (%d from the GPU)\n", sum, sum-2);
    printf("***************************************************************************\n\n");

    for(int i = 0; i < nrows; i++){
    	printf("%d ", idxs[i]);
    }
    printf("\n");

    free(data1d);
    free(idxs);


//    exit(EXIT_SUCCESS);
}
